#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <math.h>
#define PI 3.14159265359
#define a0 4-((10*pow(PI,2))/9)


double *linspace(double ini, double fin, double step,int *size);
void write_buffer(double *space, double *vals,char * file, int space_size);
__global__ void calc_term(double *xrange, double*result, int tam);
__global__ void def_funct(double *matrix, double* reconstructed_funct, int tam);
__global__ void print_grid(double *matrix, int i, int j);



int main(){
    int n_max=24;
    int tam;
    double *x_range=linspace(-3.14,3.14,.1,&tam);
    double *lins_c, *mat_rec,*reconstructed_gpu;//Apuntadores en gpu
    double *result=(double*)calloc(tam*n_max,sizeof(double));//Matriz de resultados
    double *reconstructed_funct=(double*)malloc(tam*sizeof(double));//Función reconstruida
    for (int i = 0; i < tam; i++)
    {   
        reconstructed_funct[i]=a0/2-2;
    }
    hipMalloc((void**)&lins_c,tam*sizeof(double));//Linspace
    hipMalloc((void**)&reconstructed_gpu,tam*sizeof(double));//Resultante

    hipMalloc((void**)&mat_rec,n_max * tam*sizeof(double));//Reconstruccion

    hipMemcpy(lins_c,x_range,tam*sizeof(double),hipMemcpyHostToDevice); //x_range ->lins_c
    hipMemcpy(mat_rec,result,tam*sizeof(double),hipMemcpyHostToDevice);//result ->mat_rec
    hipMemcpy(reconstructed_gpu,reconstructed_funct,tam*sizeof(double),hipMemcpyHostToDevice);// reconstructed -> result funct

    dim3 threadsPerBlock( n_max);
    dim3 numBlocks(tam);

    calc_term<<<numBlocks,threadsPerBlock>>>(lins_c,mat_rec,n_max);//Reconstruccion de la funcion
    dim3 threadsPerBlock2(tam);//longitud del linspace
    dim3 block1(1);
    def_funct<<<block1,threadsPerBlock2>>>(mat_rec,reconstructed_gpu,n_max);//FUncion resultante
    //print_grid<<<1,1>>>(mat_rec,tam,n_max);

    hipMemcpy(reconstructed_funct,reconstructed_gpu,tam*sizeof(double),hipMemcpyDeviceToHost);//reconstructed_gpu -> reconstructed
    for (int i = 0; i < tam; i++) {
            if(i==tam-1){
                printf("(%lf,%f)\n", reconstructed_funct[i],x_range[i]);
            }else{
                printf( "(%lf,%f), ", reconstructed_funct[i],x_range[i]);
            }

            
        }
    write_buffer(x_range,reconstructed_funct,"result.csv",tam);

    return 0;
}



__global__ void calc_term(double *xrange, double*result,int tam){
    int i = blockIdx.x ;//EL bloque nos dice la coordenada del linspace x
    //int m= blockIdx.x * blockDim.y + threadIdx.y;
    int n = threadIdx.x; //El thread nos dice el termino de la suma 
    n+=1;
    double x= xrange[i];

    result[threadIdx.x+tam*i]=(-20 * cos(n * PI))/(3 * pow(n,2)) * cos((n*x)); //Genera los terminos de la serie
    //printf("%f ",result[(n-1)*tam+i]);


}

__global__ void print_grid(double *matrix, int i, int j){
    for(int k=0; k<i; k++){
        for(int l=0; l<j; l++){
            printf("%f ",matrix[k*i+l]);
        }
        printf("\n");
    }
    printf("\n\n");
}


__global__ void def_funct(double *matrix, double* reconstructed_funct,int n_max){
    int n = threadIdx.x;
    double aux=0;
    for (int i = 0; i < n_max; i++)
    {
        aux+=matrix[n * n_max + i];//Reconstruye la señal paralelamente a partir de los terminos de la serie   
        }
      reconstructed_funct[n]+=aux;
}




// Arreglo uniformemente espaciado para los datos en x
double *linspace(double ini, double fin, double step, int* size){
    int aux=(int)((fin-ini)/step);
    //printf("aux: %d\n",aux);
    double *space=(double*)malloc(aux*sizeof(double)+1);
    double n_step=(fin-ini)/aux;
    if(space==NULL){
        perror("Not enough memory");
        exit(-1);
    }
    int i=0;
    space[i]=ini;
    //printf("%f, ",space[i]);
    for(int i=1; i<aux+1; i++){
        space[i]=space[i-1]+n_step;
        //printf("(%f %d), ",space[i],i);
    }
    *size=aux+1;
return space;

}

void write_buffer(double *space, double *vals,char * file, int space_size){
    FILE *f=fopen(file,"w");
    if(f==NULL){
        perror("File error");
        exit(-1);
    }
    for (int i = 0; i < space_size; i++) {
        if(i==space_size-1){
            fprintf(f, "%lf\n", space[i]);
        }else{
            fprintf(f, "%lf, ", space[i]);
        }
        
    }

    for (int i = 0; i < space_size; i++) {
        if(i==space_size-1){
            fprintf(f, "%lf\n", vals[i]);
        }else{
            fprintf(f, "%lf, ", vals[i]);
        }
        
    }
    
}