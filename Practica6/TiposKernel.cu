#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void suma_GPU(int a, int b, int *c)
{
   *c = a + b;
}

__host__ int suma_CPU(int a, int b)
{
   return (a + b);
}

int main(int argc, char** argv)
{

   int n1=1, n2=2, c=0;
   int *host_c;
   int m1=10, m2=20;
   int *dev_c;

   host_c=(int*)malloc(sizeof(int));

   hipMalloc((void**)&dev_c,sizeof(int));

   c=suma_CPU(n1, n2);
   
   printf("\nCPU: %2d + %2d = %2d \n",n1, n2, c);

   suma_GPU<<<1,1>>>(m1, m2, dev_c);

   hipMemcpy(host_c,dev_c,sizeof(int),hipMemcpyDeviceToHost );

   printf("GPU: %2d + %2d = %2d \n",m1, m2, *host_c); 

   return 0;
}
