#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void HolaDesdeGPU()
{
    printf("Hola mundo desde GPU (device o dispositivo)!\n");
}

int main(void)
{
    printf("Hola mundo desde CPU (host)!\n");

    HolaDesdeGPU<<<2,10>>>();
    
    hipDeviceSynchronize();
    
    return 0;
}

