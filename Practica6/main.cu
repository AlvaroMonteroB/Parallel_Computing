#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
int Tam=10;
void most_mat(float mat[10][10]);
void multiply_matrices(float A[10][10], float B[10][10], float *C);


__global__ void mat_mul(float *mat1, float *mat2, float *mat3,int tam);

int main(){
    srand(time(NULL));
    float mat1[10][10];
    float mat2[10][10];
    float mat3[10][10]={{0,0,0},{0,0,0},{0,0,0}};
    float mat3_fake[10][10];
    for (int i = 0; i < 10; i++)
    {
        for (int j = 0; j < 10; j++)
        {
            mat1[i][j]=rand()%10;
            mat2[i][j]=rand()%10;
        }
        
    }
    printf("Matriz 1\n");
    most_mat(mat1);
    printf("Matriz 2\n");
    most_mat(mat2);
    float *devA, *devB, *devC;
    dim3 threadsPerBlock(10, 10);
    dim3 numBlocks(1, 1);
    hipMalloc((void**)&devA,Tam*Tam*sizeof(float));
    hipMalloc((void**)&devB,Tam*Tam*sizeof(float));
    hipMalloc((void**)&devC,Tam*Tam*sizeof(float));
    
    hipMemcpy(devA,mat1,Tam*Tam*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(devB,mat2,Tam*Tam*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(devC,mat3,Tam*Tam*sizeof(int),hipMemcpyHostToDevice);

    mat_mul<<<numBlocks,threadsPerBlock>>>(devA,devB,devC,Tam);

    hipMemcpy(mat3,devC,Tam*Tam*sizeof(int),hipMemcpyDeviceToHost);

    hipFree(devA);
    hipFree(devB);
    hipFree(devC);

   hipDeviceReset();
    multiply_matrices(mat1,mat2,*mat3_fake);
    bool band=1;
    //printf("%f y %f\n",mat3[0][0],mat3_fake[0][0]);
    
    printf("Matriz cuda\n");
    most_mat(mat3);
    printf("Matriz cpu\n");
    most_mat(mat3_fake);


    for(int i = 0; i<10; i++){
        for (int j= 0; j<10; j++)
        {
            if(mat3[i][j]!=mat3_fake[i][j]){
                printf("Los resultados no son iguales\n");
                band=0;
                printf("%f y %f en (%i,%i) \n",mat3[i][j],mat3_fake[i][j],i,j);
                break;
            }
        }
        if(!band){
            break;
        }
    }

    return 0;
}





void most_mat(float mat[10][10]){
    for (int i = 0; i < 10; i++)
    {
        for (int j = 0; j < 10; j++)
        {
            printf("%f ",mat[i][j]);
        }
            printf("\n");
        
    }
    return;
}


__global__ void mat_mul(float *mat1, float *mat2, float *mat3,int tam){
    int n=/*blockIdx.x * blockDim.x + */threadIdx.x;//NUmero de columna
    int m=/*blockIdx.y * blockDim.y + */threadIdx.y;//numero de fila de mat 1 o mat 3
    float sum=0.0f;
    for (int i = 0; i < tam; i++)
    {
        sum+=mat1[m*tam+i]*mat2[i*tam+n];
    }
    mat3[n*tam+m]=sum;
}


void multiply_matrices(float A[10][10], float B[10][10], float *C) {
    // Inicializamos la matriz C con ceros
    for (int i = 0; i < 10; i++) {
        for (int j = 0; j < 10; j++) {
            C[i+j*Tam] = 0;
        }
    }
    
    // Multiplicación de matrices
    for (int i = 0; i < 10; i++) {
        for (int j = 0; j < 10; j++) {
            for (int k = 0; k < 10; k++) {
                C[i+j*Tam] += A[i][k] * B[k][j];
            }
        }
    }
}